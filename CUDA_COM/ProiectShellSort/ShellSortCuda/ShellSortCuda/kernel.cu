#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>


__global__ void ShellSortKernel( int *a, int N)  //este definit nucleul
{
    int i = threadIdx.x;   //contine indexul firului din bloc
	int j = threadIdx.y;

	int  temp;
	int dim;   //cate elemente sa imi ia

	for (dim = N / 2; dim > 0; dim /= 2)   

		for (i = dim; i < N; i++)  //procesul i imi ia ultimul element, iar procesul j imi ia primul element

			for (j = i - dim; j >= 0 && a[j]>a[j + dim]; j -= dim) {

				temp = a[j];   // daca primul element este mai mare ca al doilea mi le schimba intre ele

				a[j] = a[j + dim];

				a[j + dim] = temp;

			}
}

int main()
{
	size_t N = 10;
	int array[] = { 9, 6, 1, 2, 7, 3, 8, 4, 10, 5 };
	int  i;

	int *d_a;  //vectorul buffer alocat 
	hipMalloc(&d_a, N);  //alocam memorie liniara  // se aloca vectorul in memoria dispozitivului si dimensiunea acestuia
	
	memcpy(d_a, array, sizeof(int)*N);    //transferul de date între memoria gazdă și memoria dispozitivului
	//unde sa se puna, ce sa se puna si dimensiunea


	int numBlocks = 1;
	
	// Lansați un kernel pe GPU cu fir N pentru fiecare element.
	for (int k = 0; k < N; k++) {

		ShellSortKernel << <numBlocks, N >> >(d_a, N);
		
		//Fiecare fir care execută nucleul are un ID de thread unic, accesibil în cadrul kernelului, prin intermediul variabilei threadIdx încorporate.
		//__global__ void VecAdd(float* A, float* B, float* C, int N)
		//VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
	}

   
	for (i = 0; i < N; i++)
		printf("%d ", array[i]);

	hipFree(d_a);  //eliberam memoria liniara
    

    return 0;
}


