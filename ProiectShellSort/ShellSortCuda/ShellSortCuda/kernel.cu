#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>


__global__ void ShellSortKernel( int *a, int N)
{
    int i = threadIdx.x;
	int j = threadIdx.y;

	int  temp;
	int gap;

	for (gap = N / 2; gap > 0; gap /= 2)

		for (i = gap; i < N; i++)

			for (j = i - gap; j >= 0 && a[j]>a[j + gap]; j -= gap) {

				temp = a[j];

				a[j] = a[j + gap];

				a[j + gap] = temp;

			}
}

int main()
{
	size_t N = 10;
	int array[] = { 9, 6, 1, 2, 7, 3, 8, 4, 10, 5 };
	int  i;

	int *d_a;
	hipMalloc(&d_a, N);
	memcpy(d_a, array, sizeof(int)*N);


	int numBlocks = 1;
	
	// Launch a kernel on the GPU with N thread for each element.
	for (int k = 0; k < N; k++) {

		ShellSortKernel << <numBlocks, N >> >(d_a, N);
	}

   
	for (i = 0; i < N; i++)
		printf("%d ", array[i]);

	hipFree(d_a);
    

    return 0;
}


